#include "hip/hip_runtime.h"
#include <iostream>
#include "GPUACC.cuh"

using namespace std;

int sum_int(int a, int b);
int sum_int(int a, int b) {
	int c;
	c = a + b;
	return c;
}


int main() 
{
	int a, b, c;
	int d;
	a = 3; 
	b = 4;
	c = sum_int(3, 4);
	GPUACC gpuacc;
	gpuacc.sum_cuda(a, b, &d);
	cout << "CPU를 통한 합 : " << a << "와 " << b << "의 합은 " << c << "입니다." << endl;
	cout << "GPU를 통한 합 : " << a << "와 " << b << "의 합은 " << d << "입니다." << endl;
	system("pause");
	return 0;

}


__global__ void sum_kernel(int a, int b, int *c) 
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	c[tid] = a + b;
}

int GPUACC::sum_cuda(int a, int b, int *c) 
{
	int* f;
	hipMalloc((void**)&f, sizeof(int) * 1);
	hipMemcpy(f, c, sizeof(int) * 1, hipMemcpyHostToDevice);
	sum_kernel << <1, 1 >> > (a, b, f);
	hipMemcpy(c, f, sizeof(int) * 1, hipMemcpyDefault);
	hipFree(f);

	return true;
}