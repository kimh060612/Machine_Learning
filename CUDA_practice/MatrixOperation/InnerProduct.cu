
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <stdlib.h>
#include <stdio.h>
#define SIZE 33*1024
#define min(a, b) a > b ? b : a

const int threadperBlock = 256;
const int blocksperGrid = min(32, (SIZE + threadperBlock - 1) / threadperBlock);

__global__ void dot(float *a, float *b, float *c)
{
    __shared__ float cache[threadperBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0;
    while (tid < SIZE)
    {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    cache[cacheIndex] = temp;

    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0)
    {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

int main()
{
    
    float *a, *b, *c_, c;
    float *d_a, *d_b, *d_c_;
    a = (float *)malloc(SIZE*sizeof(float));
    b = (float *)malloc(SIZE*sizeof(float));
    c_ = (float *)malloc(blocksperGrid*sizeof(float));

    hipMalloc(&d_a, SIZE*sizeof(float));
    hipMalloc(&d_b, SIZE*sizeof(float));
    hipMalloc(&d_c_, blocksperGrid*sizeof(float));
    for (int i = 0; i < SIZE; ++i)
    {
        a[i] = i;
        b[i] = i;
    }

    hipMemcpy(d_a, a, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, SIZE*sizeof(float), hipMemcpyHostToDevice);
    
    dot<<<blocksperGrid, threadperBlock>>>(d_a, d_b, d_c_);

    hipMemcpy(c_, d_c_, blocksperGrid*sizeof(float), hipMemcpyDeviceToHost);

    c = 0;
    for (int i = 0; i < blocksperGrid; i++)
    {
        c += c_[i];
    }
    printf("%.6g\n", c);
    
    float K = 0.;
    for (int i = 0; i < SIZE; i++)
    {
        K += i*i;
    }
    printf("%.6g\n", K);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c_);
    
    free(a);
    free(b);
    free(c_);

    return 0;
}
